#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "../utils/utils.cuh"

struct Info{
    int data;
    char* text;
};

__global__ void kernel(Info* info){
    printf("[GPU] info->data = %d\n", info->data);
    printf("[GPU] info->text = %p\n", info->text);
    printf("[GPU] info->text = %s\n", info->text);
}



void launch(Info* info){
    Info* info_d;
    char* text_d;
    int text_len = strlen(info->text);

    // info_d 是host variable，但是存储的地址，指向device memory
    hipMalloc(&info_d, sizeof(Info));
    hipMalloc(&text_d, text_len);

    // info是地址，指向host memory；info_d是地址，指向device memory，所以使用hipMemcpyHostToDevice
    hipMemcpy(info_d, info, sizeof(Info), hipMemcpyHostToDevice);
    hipMemcpy(text_d, info->text, text_len, hipMemcpyHostToDevice);

    // 仅仅完成前面两步还不够，这个时候info->text和info_d->text完全一样，都指向host memory
    // 需要将text_d的值（text_d存储的地址）赋给info_d->text
    // text_d存储的是device memory address，但本身是一个host variable
    // info_d->text已经位于device memory上了
    // 因此direction选择hipMemcpyHostToDevice
    hipMemcpy(&(info_d->text), &text_d, sizeof(char*), hipMemcpyHostToDevice);

    printf("String's location in device is %p\n", text_d);

    kernel<<<1,1>>>(info_d);
    CHECK(hipDeviceSynchronize());

    hipFree(text_d);
    hipFree(info_d);

    return;

}



int main(){
    Info* info = (Info*)malloc(sizeof(Info));
    info->data = 1;
    const char* text = "Hello, world!";
    info->text = (char*)malloc(strlen(text)+1);
    strcpy(info->text, text);

    printf("[CPU] info->data = %d\n", info->data);
    printf("[CPU] info->text = %p\n", info->text);
    printf("[CPU] text's address = %p\n", text);


    launch(info);
    free(info->text);
    free(info);


    // Unified Memory
    Info* info_um = NULL;
    hipMallocManaged(&info_um, sizeof(Info));
    hipMallocManaged(&(info_um->text), strlen(text)+1);
    info_um->data = 1;
    strcpy(info_um->text, text);
    kernel<<<1,1>>>(info_um);
    CHECK(hipDeviceSynchronize());
    hipFree(info_um->text);
    hipFree(info_um);
    


    return 0;
}