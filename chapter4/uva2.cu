#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int a_d;

int main(){
    int *var, *var_d;
    size_t size = 4;
    size_t bytes = sizeof(int) * size;
    hipHostAlloc(&var, bytes, hipHostMallocMapped);
    hipHostGetDevicePointer(&var_d, var, 0);
    printf("Unified Virtual Address\n");
    printf("[CPU] %p\n", var);
    printf("[GPU] %p\n", var_d);

    int a_h = 1;
    int* a_p;
    hipGetSymbolAddress((void**)&a_p, HIP_SYMBOL(a_d));
    printf("Symbol Address\n");
    printf("[CPU] %p\n", &a_h);
    printf("[GPU] %p\n", a_p);
    printf("[Error] Use & in host to get address of device variable is %p\n", &a_d);


    hipHostFree(var);
    hipDeviceReset();
    return 0;
}