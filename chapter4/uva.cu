#include "../utils/utils.cuh"



int main(){
    int *a, *b, *c, *d;
    auto size = 1<<20;
    int byte = size * sizeof(int);
    dim3 block(256);
    dim3 grid((size-1+block.x)/block.x);
    double s,e;

    hipHostAlloc(&a, byte, hipHostMallocMapped);
    hipHostAlloc(&b, byte, hipHostMallocMapped);
    hipHostAlloc(&c, byte, hipHostMallocMapped);
    hipHostAlloc(&d, byte, hipHostMallocMapped);

    init(a, size);
    init(b, size);

    s = get_time();
    add_device<<<grid, block>>>(a,b,c,size);
    hipDeviceSynchronize();
    e = get_time();
    printf("[GPU] time = %.6f\n", e - s);


    s = get_time();
    add_host(a,b,d,size);
    e = get_time();
    printf("[CPU] time = %.6f\n", e - s);


    int flag = is_same(c,d,size);
    printf("Flag = %d\n", flag);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipHostFree(d);
    hipDeviceReset();

    return 0;



}