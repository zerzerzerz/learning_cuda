#include <stdio.h>
#include <hip/hip_runtime.h>

int main(){
    hipDeviceProp_t device_prop;
    int dev = 0;
    hipGetDeviceProperties(&device_prop, dev);
    printf("%s\n", device_prop.name);
    return 0;
}