#include "hip/hip_runtime.h"
#include "../utils/utils.cuh"

// mm means matrix multiplication
void mm(int *A, int *B, int *C, int row_A, int col_A, int col_B){
    int sum, a, b;
    int row_B = col_A;
    int row_C = row_A;
    int col_C = col_B;

    for(int i=0; i<row_A; ++i){
        for(int j=0; j<col_B; ++j){
            sum = 0;
            for(int k=0; k<col_A; ++k){
                a = A[i*col_A + k];
                b = B[k*col_B + j];
                sum += a*b;
            }
            C[i*col_C + j] = sum;
        }
    }
}


int is_same(int *A, int *B, int N, double eps=1e-6){
    for(int i=0; i<N; ++i){
        int diff = A[i] - B[i];
        if(diff < 0){
            diff = -diff;
        }
        if((double)diff > eps){
            return 0;
        }
    }
    return 1;
}


__global__ void mm_gpu(int *A, int *B, int *C, int row_A, int col_A, int col_B){
    int row_B = col_A;
    int row_C = row_A;
    int col_C = col_B;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < row_C && j < col_C){
        int sum = 0;
        int a = 0;
        int b = 0;
        for(int k=0; k<col_A; ++k){
            a = A[i*col_A+k];
            b = B[k*col_B+j];
            sum += a*b;
        }
        // printf("i=%d, j=%d, loc=%d, val=%d\n", i, j, i*col_C+j, sum);
        // printf("Global location = %d\n", i*col_C+j);
        C[i*col_C+j] = sum;
    }


}

void display(int *A, int row_A, int col_A){
    for(int i=0; i<row_A; ++i){
        for(int j=0; j<col_A; ++j){
            printf("%d ", A[i*col_A+j]);
        }
        printf("\n");
    }
}

int main(){
    int row_A = 256;
    int col_A = 512;
    int row_B = col_A;
    int col_B = 1024;
    int row_C = row_A;
    int col_C = col_B;

    int n_A = row_A * col_A;
    int n_B = row_B * col_B;
    int n_C = row_C * col_C;

    int bytes_A = n_A * sizeof(int);
    int bytes_B = n_B * sizeof(int);
    int bytes_C = n_C * sizeof(int);

    int dim1 = 32;
    int dim2 = 32;

    int *A = (int*) malloc(bytes_A);
    int *B = (int*) malloc(bytes_B);
    int *C = (int*) malloc(bytes_C);
    int *C_cpu_from_gpu = (int*) malloc(bytes_C);

    init(A, n_A);
    init(B, n_B);

    double s,e;
    s = get_time();
    mm(A,B,C,row_A,col_A,col_B);
    e = get_time();
    printf("cpu: %f sec\n", e-s);

    dim3 block(dim1, dim2);
    dim3 grid((row_C+block.x-1) / block.x, (col_C+block.y-1) / block.y);
    // printf("blockDim=(%d,%d,%d)\n", block.x, block.y, block.z);
    // printf("gridDim=(%d,%d,%d)\n", grid.x, grid.y, grid.z);


    int *A_gpu, *B_gpu, *C_gpu;
    hipMalloc(&A_gpu, bytes_A);
    hipMalloc(&B_gpu, bytes_B);
    hipMalloc(&C_gpu, bytes_C);

    hipMemcpy(A_gpu, A, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, bytes_B, hipMemcpyHostToDevice);

    s = get_time();
    mm_gpu<<<grid, block>>>(A_gpu,B_gpu,C_gpu,row_A,col_A,col_B);
    hipDeviceSynchronize();
    e = get_time();
    printf("gpu: %f sec\n", e-s);

    hipMemcpy(C_cpu_from_gpu, C_gpu, bytes_C, hipMemcpyDeviceToHost);
    
    printf("%d\n", is_same(C, C_cpu_from_gpu, n_C));

    // display(C, row_C, col_C);
    // printf("\n");
    // display(C_cpu_from_gpu, row_C, col_C);

    free(A);
    free(B);
    free(C);
    free(C_cpu_from_gpu);
    hipDeviceReset();

    return 0;
}
